#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <vector>
#include <string>
#include <filesystem>
#include <fstream>
#include <ctime>

using namespace cv;
namespace fs = std::filesystem;

// CUDA kernel for 2D Gaussian blur (simplified 3x3 kernel)
__global__ void gaussianBlurKernel(unsigned char* input, unsigned char* output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float kernel[3][3] = {
        {1.0/16, 2.0/16, 1.0/16},
        {2.0/16, 4.0/16, 2.0/16},
        {1.0/16, 2.0/16, 1.0/16}
    };

    for (int c = 0; c < channels; c++) {
        float sum = 0.0;
        for (int ky = -1; ky <= 1; ky++) {
            for (int kx = -1; kx <= 1; kx++) {
                int px = min(max(x + kx, 0), width - 1);
                int py = min(max(y + ky, 0), height - 1);
                sum += input[(py * width + px) * channels + c] * kernel[ky + 1][kx + 1];
            }
        }
        output[(y * width + x) * channels + c] = (unsigned char)sum;
    }
}

// Get current timestamp for logging
std::string getTimestamp() {
    time_t now = time(0);
    char* dt = ctime(&now);
    std::string timestamp(dt);
    return timestamp.substr(0, timestamp.length() - 1); // Remove newline
}

// Process a batch of images and log results
void processImages(const std::string& inputDir, const std::string& outputDir, std::ofstream& logFile) {
    std::vector<std::string> imageFiles;
    for (const auto& entry : fs::directory_iterator(inputDir)) {
        if (entry.path().extension() == ".jpg" || entry.path().extension() == ".png") {
            imageFiles.push_back(entry.path().string());
        }
    }

    if (imageFiles.empty()) {
        std::cerr << "No images found in " << inputDir << std::endl;
        logFile << "[" << getTimestamp() << "] ERROR: No images found in " << inputDir << std::endl;
        return;
    }

    logFile << "[" << getTimestamp() << "] INFO: Starting batch processing of " << imageFiles.size() << " images" << std::endl;

    for (const auto& file : imageFiles) {
        // Load image
        Mat img = imread(file, IMREAD_COLOR);
        if (img.empty()) {
            std::cerr << "Failed to load " << file << std::endl;
            logFile << "[" << getTimestamp() << "] ERROR: Failed to load " << file << std::endl;
            continue;
        }

        int width = img.cols;
        int height = img.rows;
        int channels = img.channels();
        size_t size = width * height * channels;

        // Allocate device memory
        unsigned char *d_input, *d_output;
        hipMalloc(&d_input, size);
        hipMalloc(&d_output, size);

        // Copy image data to device
        hipMemcpy(d_input, img.data, size, hipMemcpyHostToDevice);

        // Launch kernel
        dim3 blockSize(16, 16);
        dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
        gaussianBlurKernel<<<gridSize, blockSize>>>(d_input, d_output, width, height, channels);

        // Copy result back to host
        Mat outputImg(height, width, CV_8UC3);
        hipMemcpy(outputImg.data, d_output, size, hipMemcpyDeviceToHost);

        // Save output
        std::string outputFile = outputDir + "/" + fs::path(file).filename().string();
        imwrite(outputFile, outputImg);

        std::cout << "Processed: " << file << " -> " << outputFile << std::endl;
        logFile << "[" << getTimestamp() << "] INFO: Processed " << file << " -> " << outputFile 
                << " (Size: " << width << "x" << height << ")" << std::endl;

        // Clean up
        hipFree(d_input);
        hipFree(d_output);
    }

    logFile << "[" << getTimestamp() << "] INFO: Batch processing completed" << std::endl;
}

int main(int argc, char** argv) {
    if (argc != 5 || std::string(argv[1]) != "--input_dir" || std::string(argv[3]) != "--output_dir") {
        std::cerr << "Usage: " << argv[0] << " --input_dir <path> --output_dir <path>" << std::endl;
        return -1;
    }

    std::string inputDir = argv[2];
    std::string outputDir = argv[4];

    if (!fs::exists(inputDir) || !fs::exists(outputDir)) {
        std::cerr << "Input or output directory does not exist!" << std::endl;
        return -1;
    }

    // Open log file
    std::ofstream logFile("processing_log.txt");
    if (!logFile.is_open()) {
        std::cerr << "Failed to open log file!" << std::endl;
        return -1;
    }

    processImages(inputDir, outputDir, logFile);
    logFile.close();
    return 0;
}
